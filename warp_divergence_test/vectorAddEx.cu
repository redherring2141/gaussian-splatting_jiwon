#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n, double max_limit, int n_iter)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    for (int iter = 0; iter < n_iter; iter++)
    {
        if (id < n)
        {
            if (c[id] < max_limit)
            {
                c[id] = a[id] + b[id];
            }
        }
    }
}
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n_x = 4;
    int n_y = 4;
    int n = n_x*n_y;
    double portion = 0.5;
    double max_limit = 100;
    int outer_iter = int(max_limit);
    int inner_iter = int(max_limit);
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    for (int iter=0; iter<int(max_limit); iter++)
    {
        // Initialize vectors on host
        for(int idx = 0; idx < n; idx++ )
        {
            //h_a[i] = sin(i)*sin(i);
            //h_b[i] = cos(i)*cos(i);
            h_a[idx] = (idx%2)==0 ? 0 : (max_limit*portion);
            h_b[idx] = 1;
        }

        // Copy host vectors to device
        hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
        hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
    
        int blockSize, gridSize;
    
        // Number of threads in each thread block
        blockSize = 1024;
    
        // Number of thread blocks in grid
        gridSize = (int)ceil((float)n/blockSize);
    
        // Execute the kernel
        vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n, max_limit, n_iter);
    
        // Copy array back to host
        hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
    }
 
    // Sum up vector c and print result divided by n, this should equal 1 within error

    printf("<Final result>\n");
    for(int i=0; i<n; i++)
    {
        for(int j=0; j<n; j++)
        {
            printf("%d ");
        }
        printf("\n");
    }
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}