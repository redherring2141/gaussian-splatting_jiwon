#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

#include "nvToolsExt.h"	//JWLB_20240112

torch::Tensor
distCUDA2(const torch::Tensor& points)
{
	nvtxRangePush("[JWLB-spatial.cu-distCUDA2]00_1distCUDA2");//JWLB_20240112
	hipEvent_t start_JWLB, stop_JWLB; float msec=0; hipEventCreate(&start_JWLB);	hipEventCreate(&stop_JWLB);	//JWLB_20240112
	hipEventRecord(start_JWLB);	hipEventSynchronize(start_JWLB);	//JWLB_20240112

  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor means = torch::full({P}, 0.0, float_opts);
  
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

	hipEventRecord(stop_JWLB);		hipEventSynchronize(stop_JWLB);	hipEventElapsedTime(&msec, start_JWLB, stop_JWLB);	//JWLB_20240112
	std::cout << "[JWLB-spatial.cu-distCUDA2]00_1distCUDA2: " << msec << "ms" << std::endl; //JWLB_20240112
	nvtxRangePop();//JWLB_20240112

  return means;
}